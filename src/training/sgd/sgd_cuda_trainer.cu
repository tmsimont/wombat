// Copyright 2017 Trevor Simonton

#include "src/sgd_trainers/sgd_cuda_trainer.h"

float *d_Wih, *d_Woh, *d_expTable;

SGDCUDATrainer::SGDCUDATrainer(int num_batches, int batch_size) {
  this->batch_size = batch_size;
  this->num_batches = num_batches;

  if (hs) {
    twords_batch_size = MAX_CODE_LENGTH;
  } else {
    twords_batch_size = negative + 1;
  }
  labels_batch_size = twords_batch_size;

  cwords_batch_size = (2 * window + 1);

  int data_batch_size = cwords_batch_size * twords_batch_size;

  const long bytes_needed =
    (long)num_batches
    * (long)batch_size
    * (long)data_batch_size
    * sizeof(int);

  twords_bytes = bytes_needed;
  labels_bytes = bytes_needed;
  cwords_bytes = bytes_needed;

  checkCuda(hipHostMalloc(reinterpret_cast<void **>(&cwords), cwords_bytes));
  checkCuda(hipHostMalloc(reinterpret_cast<void **>(&twords), twords_bytes));
  checkCuda(hipHostMalloc(reinterpret_cast<void **>(&labels), labels_bytes));

  hipMalloc(reinterpret_cast<void **>(&d_cwords), cwords_bytes);
  hipMalloc(reinterpret_cast<void **>(&d_twords), twords_bytes);
  hipMalloc(reinterpret_cast<void **>(&d_labels), labels_bytes);

  num_streams = num_batches;

  streams = reinterpret_cast<hipStream_t *>(
      malloc(num_streams * sizeof(hipStream_t)));
  memoutEvents = reinterpret_cast<hipEvent_t *>(
      malloc(num_streams * sizeof(hipEvent_t)));

  for (int i = 0; i < num_streams; i++) {
    hipStreamCreate(&(streams[i]));
    hipEventCreate(&(memoutEvents[i]));
  }
}

SGDCUDATrainer::~SGDCUDATrainer() {
  hipHostFree(cwords);
  hipHostFree(twords);
  hipHostFree(labels);

  hipFree(d_cwords);
  hipFree(d_twords);
  hipFree(d_labels);
}

void SGDCUDATrainer::memtoCUDA() {
  int twi = 0;
  int cwi = 0;
  for (int i = 0; i < wombat.size(); i++) {
    wombat[i]->copyTWords(twords + twi);
    wombat[i]->copyLabels(labels + twi);
    wombat[i]->copyCWords(cwords + cwi);
    twi += wombat[i]->numTWords();
    cwi += wombat[i]->numCWords();
  }

  for (int i = 0; i < wovbat.size(); i++) {
    twords[twi] = wovbat[i].tword;
    labels[twi] = wovbat[i].label;
    cwords[cwi] = wovbat[i].cword;
    twi++;
    cwi++;
  }

  checkCuda(hipMemcpyAsync(
        d_cwords,
        cwords,
        cwords_bytes,
        hipMemcpyHostToDevice,
        streams[0]));
  checkCuda(hipMemcpyAsync(
        d_twords,
        twords,
        twords_bytes,
        hipMemcpyHostToDevice,
        streams[0]));
  checkCuda(hipMemcpyAsync(
        d_labels,
        labels,
        labels_bytes,
        hipMemcpyHostToDevice,
        streams[0]));
  checkCuda(hipEventRecord(memoutEvents[0], streams[0]));
}

void SGDCUDATrainer::train() {
  if (loaded_sets < (batch_size * num_batches)) {
    return;
  }

  memtoCUDA();

  CallKernels(hs, wombat.size(), wovbat.size(), streams + 0,
    d_Wih,
    d_Woh,
    d_cwords,
    0,
    d_twords,
    0,
    d_labels,
    labels_batch_size,
    hidden_size,
    alpha,
    MAX_EXP);
}

void SGDCUDATrainer::clear() {
  SGDBatchTrainer::clear();
  for (auto &p : wombat) {
    delete p;
  }
  wombat.clear();
  wovbat.clear();
  used.clear();
}

void SGDCUDATrainer::loadSet(TCBufferReader *tc_reader) {
  if (loaded_sets == batch_size * num_batches) return;
  if (loaded_sets == 0)
    hipEventSynchronize(memoutEvents[0]);


  int target_indices[twords_batch_size];
  int labels[twords_batch_size];
  int targets_to_load = 0;
  if (hs) {
    int target = tc_reader->targetWord();
    for (int k = 0; k < vocab[target].codelen; k++) {
          target_indices[targets_to_load] = vocab[target].point[k];
          labels[targets_to_load] = vocab[target].code[k];
          targets_to_load++;
    }
  } else {
    for (int i = 0; i < twords_batch_size; ++i) {
      if (i == 0) {
        target_indices[i] = tc_reader->targetWord();
        labels[i] = 1;
      } else {
        int sample = 0;
        next_random = next_random * (unsigned long long) 25214903917 + 11;
        sample = table[(next_random >> 16) % table_size];
        if (!sample)
          sample = next_random % (vocab_size - 1) + 1;
        target_indices[i] = sample;
        labels[i] = 0;
      }
      targets_to_load++;
    }
  }

  int twi = 0;
  while (targets_to_load-twi > 0) {
    int numcw = tc_reader->numCWords();
    int cwi = 0;
    while (numcw > 0) {
      if (numcw >= 8 && (targets_to_load - twi) >= 4) {
        MOP4x8 *m =  new MOP4x8();
        for (int i = 0; i < 8; i++) {
          m->addCWord(*(tc_reader->cwords() + cwi++));
          numcw--;
        }
        for (int i = 0; twi+i < targets_to_load && i < 4; i++) {
          m->addTWord(target_indices[twi+i], labels[twi+i]);
        }
        wombat.push_back(m);
      } else {
        for (int i = 0; twi+i < targets_to_load && i < 4; i++) {
          VOP v(
              target_indices[twi+i],
              *(tc_reader->cwords() + cwi),
              labels[twi+i]);
          wovbat.push_back(v);
        }
        cwi++;
        numcw--;
      }
    }
    twi += 4;
  }

  loaded_sets++;
}

void InitNetCUDA(real **Wih, real **Woh) {
  checkCuda(hipMalloc((void **)&d_Woh,
        (long long)vocab_size * hidden_size * sizeof(float)));
  checkCuda(hipMemcpy(d_Woh, *Woh,
        (long long)vocab_size * hidden_size * sizeof(float),
        hipMemcpyHostToDevice));
  checkCuda(hipMalloc((void **)&d_Wih,
        (long long)vocab_size * hidden_size * sizeof(float)));
  checkCuda(hipMemcpy(d_Wih, *Wih,
        (long long)vocab_size * hidden_size * sizeof(float),
        hipMemcpyHostToDevice));
}

void InitExpCUDA() {
  checkCuda(hipMalloc(reinterpret_cast<void **>(&d_expTable),
        (EXP_TABLE_SIZE + 1) * sizeof(float)));
  checkCuda(hipMemcpy(d_expTable, expTable,
        (EXP_TABLE_SIZE + 1) * sizeof(float),
        hipMemcpyHostToDevice));
}

void WiToHost(real **Wih) {
  hipDeviceSynchronize();
  checkCuda(hipMemcpy(*Wih, d_Wih,
        (long long)vocab_size * hidden_size * sizeof(float),
        hipMemcpyDeviceToHost));
}

void WoToHost(real **Woh) {
  hipDeviceSynchronize();
  checkCuda(hipMemcpy(*Woh, d_Woh,
        (long long)vocab_size * hidden_size * sizeof(float),
        hipMemcpyDeviceToHost));
}
