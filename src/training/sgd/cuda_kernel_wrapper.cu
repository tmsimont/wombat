// Copyright 2017 Trevor Simonton

#include <vector>
#include "src/sgd_trainers/cuda_kernel.wombat.cu"

void CallKernels(int hs, int wombat_size, int wovbat_size, hipStream_t* stream,
    float *d_Wih,
    float *d_Woh,
    int *d_cwords,
    int bwords_start_idx,
    int *d_twords,
    int awords_start_idx,
    int *d_labels,
    int labels_batch_size,
    int hidden_size,
    float alpha,
    int max_exp) {

  dim3 wombatBlock(8, 4);
  if (wombat_size > 0) {
    Wombat4x8<<<wombat_size, wombatBlock,
      (hidden_size * 32) * sizeof(float), *stream>>>(
        d_Wih,
        d_Woh,
        d_cwords,
        0,
        d_twords,
        0,
        d_labels,
        hidden_size,
        alpha,
        max_exp,
        hs);
  }

  dim3 vectorBlock(32, 1);
  if (wovbat_size > 0) {
    VectorTrain<<<wovbat_size, vectorBlock,
      (hidden_size * 2) * sizeof(float), *stream>>>(
        d_Wih,
        d_Woh,
        d_cwords,
        wombat_size * 8,
        d_twords,
        wombat_size * 4,
        d_labels,
        hidden_size,
        alpha,
        max_exp,
        hidden_size,
        hs);
  }
}
