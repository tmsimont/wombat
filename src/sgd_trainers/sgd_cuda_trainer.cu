// Copyright 2017 Trevor Simonton

#include "src/sgd_trainers/sgd_cuda_trainer.h"

float *d_Wih, *d_Woh, *d_expTable;

SGDCUDATrainer::SGDCUDATrainer(int num_batches, int batch_size) {
  this->batch_size = batch_size;
  this->num_batches = num_batches;

  if (hs) {
    twords_batch_size = MAX_CODE_LENGTH;
  } else {
    twords_batch_size = negative + 1;
  }
  labels_batch_size = twords_batch_size;

  cwords_batch_size = (2 * window + 1);

  int data_batch_size = cwords_batch_size * twords_batch_size;

  const long bytes_needed =
    (long)num_batches
    * (long)batch_size
    * (long)data_batch_size
    * sizeof(int);

  twords_bytes = bytes_needed;
  labels_bytes = bytes_needed;
  cwords_bytes = bytes_needed;

  checkCuda(hipHostMalloc(reinterpret_cast<void **>(&cwords), cwords_bytes));
  checkCuda(hipHostMalloc(reinterpret_cast<void **>(&twords), twords_bytes));
  checkCuda(hipHostMalloc(reinterpret_cast<void **>(&labels), labels_bytes));

  hipMalloc(reinterpret_cast<void **>(&d_cwords), cwords_bytes);
  hipMalloc(reinterpret_cast<void **>(&d_twords), twords_bytes);
  hipMalloc(reinterpret_cast<void **>(&d_labels), labels_bytes);

  // I think this idea is dead and we always use stream[0]...
  num_streams = num_batches;

  streams = reinterpret_cast<hipStream_t *>(
      malloc(num_streams * sizeof(hipStream_t)));
  memoutEvents = reinterpret_cast<hipEvent_t *>(
      malloc(num_streams * sizeof(hipEvent_t)));

  for (int i = 0; i < num_streams; i++) {
    hipStreamCreate(&(streams[i]));
    hipEventCreate(&(memoutEvents[i]));
  }
}

SGDCUDATrainer::~SGDCUDATrainer() {
  hipHostFree(cwords);
  hipHostFree(twords);
  hipHostFree(labels);

  hipFree(d_cwords);
  hipFree(d_twords);
  hipFree(d_labels);
}

void SGDCUDATrainer::memtoCUDA() {
  int twi = 0;
  int cwi = 0;
  for (int i = 0; i < wombat.size(); i++) {
    wombat[i]->copyTWords(twords + twi);
    wombat[i]->copyLabels(labels + twi);
    wombat[i]->copyCWords(cwords + cwi);
    twi += wombat[i]->numTWords();
    cwi += wombat[i]->numCWords();
  }

  for (int i = 0; i < wovbat.size(); i++) {
    twords[twi] = wovbat[i].tword;
    labels[twi] = wovbat[i].label;
    cwords[cwi] = wovbat[i].cword;
    twi++;
    cwi++;
  }

  checkCuda(hipMemcpyAsync(
        d_cwords,
        cwords,
        cwords_bytes,
        hipMemcpyHostToDevice,
        streams[0]));
  checkCuda(hipMemcpyAsync(
        d_twords,
        twords,
        twords_bytes,
        hipMemcpyHostToDevice,
        streams[0]));
  checkCuda(hipMemcpyAsync(
        d_labels,
        labels,
        labels_bytes,
        hipMemcpyHostToDevice,
        streams[0]));
  checkCuda(hipEventRecord(memoutEvents[0], streams[0]));
}

void SGDCUDATrainer::train() {
  if (loaded_sets < (batch_size * num_batches)) {
    return;
  }

  memtoCUDA();

  CallKernels(hs, wombat.size(), wovbat.size(), streams + 0, // here's the always use 0 part..
    d_Wih,
    d_Woh,
    d_cwords,
    0,
    d_twords,
    0,
    d_labels,
    labels_batch_size,
    hidden_size,
    alpha,
    MAX_EXP);
}

void SGDCUDATrainer::clear() {
  SGDBatchTrainer::clear();
  for (auto &p : wombat) {
    delete p;
  }
  wombat.clear();
  wovbat.clear();
  used.clear();
}

/**
 * I started calling things "sets" because the batch of batch shit is confusing AF.
 * I never got around to cleaning this up until now... The "set" is a bunch of "minibatches".
 * Check out the commit history on this note. I'm adding a bunch of notes in one big commit and
 * otherwise I'm not working on the master branch anymore. I'm starting to refactor things in the v2 branch.
 */
void SGDCUDATrainer::loadSet(TCBufferReader *tc_reader) {
  if (loaded_sets == batch_size * num_batches) return;
  if (loaded_sets == 0)
    hipEventSynchronize(memoutEvents[0]);


  int target_indices[twords_batch_size];
  int labels[twords_batch_size];
  int targets_to_load = 0;
  if (hs) {
    int target = tc_reader->targetWord();
    for (int k = 0; k < vocab[target].codelen; k++) {
          target_indices[targets_to_load] = vocab[target].point[k];
          labels[targets_to_load] = vocab[target].code[k];
          targets_to_load++;
    }
  } else {
    for (int i = 0; i < twords_batch_size; ++i) {
      if (i == 0) {
        target_indices[i] = tc_reader->targetWord();
        labels[i] = 1;
      } else {
        int sample = 0;
        next_random = next_random * (unsigned long long) 25214903917 + 11;
        sample = table[(next_random >> 16) % table_size];
        if (!sample)
          sample = next_random % (vocab_size - 1) + 1;
        target_indices[i] = sample;
        labels[i] = 0;
      }
      targets_to_load++;
    }
  }

  /**
   * This craziness maps all of the "minibatches" in our big batch of batches to a bunch of sub-operations.
   * We look for "minibatches" that have 4 input rows (e.g. target words / h-softmax nodes) and 8 context words
   * aka "output layers" in the minibatch.
   * We batch up all the 4x8 minibatches into MOP structures, which can then be crunched with a highly optimized
   * kernel.
   * The rest of our training minibatches get split up into VOP structures, where we use an optimized dot-product-based
   * kernel to take care of the non-4x8-minibatches.
   * We could take even more shapes here if we wanted to (e.g. 32x1, 8x4, 2x16) and write optimized kernels for them,
   * too, but i ran out of time while working on the thesis :P
   * The numbers are centered around 32 because of the way warp shuffling works in cuda...
   */
  int twi = 0;
  while (targets_to_load-twi > 0) {
    int numcw = tc_reader->numCWords();
    int cwi = 0;
    while (numcw > 0) {
      if (numcw >= 8 && (targets_to_load - twi) >= 4) {
        MOP4x8 *m =  new MOP4x8();
        for (int i = 0; i < 8; i++) {
          m->addCWord(*(tc_reader->cwords() + cwi++));
          numcw--;
        }
        for (int i = 0; twi+i < targets_to_load && i < 4; i++) {
          m->addTWord(target_indices[twi+i], labels[twi+i]);
        }
        wombat.push_back(m);
      } else {
        for (int i = 0; twi+i < targets_to_load && i < 4; i++) {
          VOP v(
              target_indices[twi+i],
              *(tc_reader->cwords() + cwi),
              labels[twi+i]);
          wovbat.push_back(v);
        }
        cwi++;
        numcw--;
      }
    }
    twi += 4;
  }

  loaded_sets++;
}

void InitNetCUDA(real **Wih, real **Woh) {
  checkCuda(hipMalloc((void **)&d_Woh,
        (long long)vocab_size * hidden_size * sizeof(float)));
  checkCuda(hipMemcpy(d_Woh, *Woh,
        (long long)vocab_size * hidden_size * sizeof(float),
        hipMemcpyHostToDevice));
  checkCuda(hipMalloc((void **)&d_Wih,
        (long long)vocab_size * hidden_size * sizeof(float)));
  checkCuda(hipMemcpy(d_Wih, *Wih,
        (long long)vocab_size * hidden_size * sizeof(float),
        hipMemcpyHostToDevice));
}

void InitExpCUDA() {
  checkCuda(hipMalloc(reinterpret_cast<void **>(&d_expTable),
        (EXP_TABLE_SIZE + 1) * sizeof(float)));
  checkCuda(hipMemcpy(d_expTable, expTable,
        (EXP_TABLE_SIZE + 1) * sizeof(float),
        hipMemcpyHostToDevice));
}

void WiToHost(real **Wih) {
  hipDeviceSynchronize();
  checkCuda(hipMemcpy(*Wih, d_Wih,
        (long long)vocab_size * hidden_size * sizeof(float),
        hipMemcpyDeviceToHost));
}

void WoToHost(real **Woh) {
  hipDeviceSynchronize();
  checkCuda(hipMemcpy(*Woh, d_Woh,
        (long long)vocab_size * hidden_size * sizeof(float),
        hipMemcpyDeviceToHost));
}
