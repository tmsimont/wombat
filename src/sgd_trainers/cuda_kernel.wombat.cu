// Copyright 2017 Trevor Simonton


#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

__global__ void Wombat4x8(
    float *Wb,
    float *Wa,
    int *bwords,
    int bwords_start_idx,
    int *awords,
    int awords_start_idx,
    int *labels,
    int hidden_size,
    float alpha,
    int max_exp,
    int hs) {

  int row = threadIdx.y;
  int col = threadIdx.x;
  int batch_index = blockIdx.x;
  int a = 4;
  int b = 8;
  int awords_index = awords_start_idx + batch_index*a;
  int labels_index = awords_start_idx + batch_index*a;
  int bwords_index = bwords_start_idx + batch_index*b;

  extern __shared__ float sw[];
  float *As = &sw[0];
  float *Bs  = &sw[4 * hidden_size];

  // load in local sets of word vectors into As and Bs
  for (int i = 0; i < hidden_size; i += b) {
    if ((i+col) < hidden_size)
      As[(hidden_size*row) + (i+col)] =
        Wa[(hidden_size * awords[awords_index + row]) + (i+col)];
  }
  for (int i = 0; i < hidden_size; i += a) {
    if ((i+row) < hidden_size)
      Bs[(hidden_size*col) + (i+row)] =
        Wb[(hidden_size * bwords[bwords_index + col]) + (i+row)];
  }

  __syncthreads();

  // activate loaded vectors into Cs
  float f = 0;
  for (int i = 0; i < hidden_size; ++i) {
    f += As[(hidden_size*row) + i] * Bs[col*hidden_size + i];
  }
  if (hs == 1) {
    if (f >= max_exp) {
      f = 0;
    } else if (f <= -max_exp) {
      f = 0;
    } else {
      f = exp(f);
      f = f / (1.0f + f);
      f = (1.0f - labels[labels_index + row] - f) * alpha;
    }
  } else {
    if (f > max_exp) {
      f = (labels[labels_index + row] - 1) * alpha;
    } else if (f < -max_exp) {
      f = labels[labels_index + row] * alpha;
    } else {
      f = exp(f);
      f = f / (1.0f + f);
      f = (labels[labels_index + row] - f) * alpha;
    }
  }

  for (int i = 0; i < hidden_size; i++) {
    // calculate local update for this thread
    float uA = f * Bs[col*hidden_size + i];
    float uB = f * As[row*hidden_size + i];

    // update column of B
    uB += __shfl_down(uB, 16);
    uB += __shfl_down(uB, 8);
    if (row == 0) {
      atomicAdd(
          Wb + (hidden_size * bwords[bwords_index + col]) + i,
          uB);
    }

    // update column of A
    uA += __shfl_down(uA, 4, 8);
    uA += __shfl_down(uA, 2, 8);
    uA += __shfl_down(uA, 1, 8);
    if (col == 0) {
      atomicAdd(
          Wa + (hidden_size * awords[awords_index + row]) + i,
          uA);
    }
  }
}

__global__ void VectorTrain(
    float *Wb,
    float *Wa,
    int *bwords,
    int bwords_start_idx,
    int *awords,
    int awords_start_idx,
    int *labels,
    int hidden_size,
    float alpha,
    int max_exp,
    int B_start,
    int hs) {

  int batch_index = blockIdx.x;
  int awords_index = awords_start_idx + batch_index;
  int labels_index = awords_start_idx + batch_index;
  int bwords_index = bwords_start_idx + batch_index;

  extern __shared__ float sv[];
  float *A1s = &sv[0];
  float *Bs  = &sv[B_start];

  float f = 0;
  for (int i = 0; i < hidden_size / 32; i++) {
    A1s[i+threadIdx.x*hidden_size/32] =
      Wa[(hidden_size * awords[awords_index]) + i + threadIdx.x*hidden_size/32];
    Bs[i+threadIdx.x*hidden_size/32] =
      Wb[(hidden_size * bwords[bwords_index]) + i + threadIdx.x*hidden_size/32];
  }

  __syncthreads();

  for (int i = 0; i < hidden_size / 32; i++) {
    f += A1s[i + threadIdx.x*hidden_size/32]
      * Bs[i + threadIdx.x*hidden_size/32];
  }
  #pragma unroll
  for (int i = 16; i > 0; i /= 2) {
    f += __shfl_down(f, i);
  }
  if (threadIdx.x == 0) {
    if (hs == 1) {
      if (f >= max_exp) {
        f = 0;
      } else if (f <= -max_exp) {
        f = 0;
      } else {
        f = exp(f);
        f = f / (1.0f + f);
        f = (1.0f - labels[labels_index] - f) * alpha;
      }
    } else {
      if (f > max_exp) {
        f = (labels[labels_index] - 1) * alpha;
      } else if (f < -max_exp) {
        f = labels[labels_index] * alpha;
      } else {
        f = exp(f);
        f = f / (1.0f + f);
        f = (labels[labels_index] - f) * alpha;
      }
    }
  }

  f = __shfl(f, 0);


  // Calculate and apply updates
  for (int i = 0; i < hidden_size/32; i++) {
    atomicAdd(
        Wa + (hidden_size * awords[awords_index])
          + i+threadIdx.x*hidden_size/32,
        f * Bs[i+threadIdx.x*hidden_size/32]);
    atomicAdd(
        Wb + (hidden_size * bwords[bwords_index])
          + i+threadIdx.x*hidden_size/32,
        f * A1s[i+threadIdx.x*hidden_size/32]);
  }
}
